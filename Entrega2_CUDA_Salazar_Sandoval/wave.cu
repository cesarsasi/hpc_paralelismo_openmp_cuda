
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <ctype.h>
#include <math.h>

#include <getopt.h>

#include <sys/time.h>
#include <time.h>

// FUNCIÓN: validar_entrada.
// ENTRADA: Parametros de entrada con valor entero a excepcion de archivoSalida.
// PROCESAMIENTO: Valida los parametros de entrada, con tal de asegurar un correcto input de variables.
// SALIDA: Valor entero, con posibilidad de "0" si son entradas invalidas y un "1" en caso contrario.
int validar_entrada(int tamanoGrila, int numPasos, int tamanoBloqueX, int tamanoBloqueY){
    int valido = 1;
    
    if(tamanoGrila == 0){
        printf("Ingrese un número de grillas válido.\n");
        valido = 0; 
    }
    if(numPasos == 0){
        printf("Ingrese un número de pasos válido.\n");
        valido = 0;
    }
    if(tamanoBloqueX == 0){
        printf("Ingrese un tamano bloque X válido.\n");
        valido = 0;
    }
    if(tamanoBloqueY == 0){
        printf("Ingrese un tamano bloque Y válido.\n");
        valido = 0;
    }
    return valido;
}

// FUNCIÓN: Matriz_ceros.
// ENTRADA: Matriz inicializada
// PROCESAMIENTO: Rellena toda la matriz con ceros.
// SALIDA: Matriz de ceros.
void matriz_ceros(float *matriz, int N){
    for(int i=0; i<N; i++){
        for(int j=0; j<N; j++){
            matriz[i*N+j] = 0.0;
        }
    }
}

// FUNCIÓN: Ejecutar_schroedinger_t0cuda
// ENTRADA: Matriz tiempo 0, largo del arreglo.
// PROCESAMIENTO: Calcula primer pulso de forma paralela.
// SALIDA: - .
__global__ void ejecutar_schroedinger_t0cuda(float *matriz_t0, int N){
	int i, j;
	i = blockDim.x*blockIdx.x + threadIdx.x;  // global index x (horizontal)
	j = blockDim.y*blockIdx.y + threadIdx.y;  // global index y (vertical)

	if((i > 0.4*N && i <0.6*N) && (j > 0.4*N && j < 0.6*N)){
        matriz_t0[i*N+j] = 20.0;
	}
}
 
// FUNCIÓN: Ejecutar_schroedinger_t1cuda
// ENTRADA: Matriz tiempo 0, Matriz tiempo 1, largo del arreglo.
// PROCESAMIENTO: Calcula tiempo 1 de forma paralela.
// SALIDA: - .
__global__ void ejecutar_schroedinger_t1cuda(float *matriz_t0, float *matriz_t1, int N){
	int i, j;
    float c = 1.0, dt = 0.1, dd = 2.0;
    float iInf, iSup, jInf, jSup, ij_t1, cola;
	i = blockDim.x*blockIdx.x + threadIdx.x;  // global index x (horizontal)
	j = blockDim.y*blockIdx.y + threadIdx.y;  // global index y (vertical)

    if(i >= 1 && i < (N-1) && j >= 1 && j < (N-1)){
        iInf = matriz_t0[(i-1)*N+j];
        iSup = matriz_t0[(i+1)*N+j];
        jInf = matriz_t0[i*N+(j-1)];
        jSup = matriz_t0[i*N+(j+1)];
        ij_t1 = matriz_t0[i*N+j];
        cola = ((dt*dt)/(dd*dd)) * (iInf+iSup+jSup+jInf-(4*ij_t1));
        matriz_t1[i*N+j] = (ij_t1 + (((c*c)/2)*cola));
	}              
}

// FUNCIÓN: Ejecutar_schroedinger_tncuda
// ENTRADA: Matriz tiempo 0, Matriz tiempo 1, Matriz tiempo actual,largo del arreglo.
// PROCESAMIENTO: Calcula tiempo n de forma paralela.
// SALIDA: - .
__global__ void ejecutar_schroedinger_tncuda(float *matriz_t0, float *matriz_t1, float *matriz_rs, int N){
	int i, j;
    float c = 1.0, dt = 0.1, dd = 2.0;
    float iInf, iSup, jInf, jSup, ij_t1, ij_t0, cola;
	i = blockDim.x*blockIdx.x + threadIdx.x;  // global index x (horizontal)
	j = blockDim.y*blockIdx.y + threadIdx.y;  // global index y (vertical)

    if(i >= 1 && i < (N-1) && j >= 1 && j < (N-1)){
        iInf = matriz_t1[(i-1)*N+j];
        iSup = matriz_t1[(i+1)*N+j];
        jInf = matriz_t1[i*N+(j-1)];
        jSup = matriz_t1[i*N+(j+1)];
        ij_t1 = matriz_t1[i*N+j];
        ij_t0 = matriz_t0[i*N+j];
        cola = ((dt*dt)/(dd*dd)) * (iInf+iSup+jSup+jInf-(4*ij_t1));
        matriz_rs[i*N+j] = ((2*ij_t1) - ij_t0 + ((c*c)*cola));
	}                 
}

// FUNCIÓN: Copiar_matriz_cuda
// ENTRADA: Matriz tiempo 0, Matriz tiempo 1, Matriz tiempo actual,largo del arreglo.
// PROCESAMIENTO: Copia los valores de las matrices a la matriz del tiempo anterior tn -> tn-1 -> tn-2.
// SALIDA: - .
__global__ void copiar_matriz_cuda(float *matriz_t0, float *matriz_t1, float *matriz_rs, int N){
    int i, j;
	i = blockDim.x*blockIdx.x + threadIdx.x;  // global index x (horizontal)
	j = blockDim.y*blockIdx.y + threadIdx.y;  // global index y (vertical)

    if(i >= 1 && i < N-1 && j >= 1 && j < N-1){
        matriz_t0[i*N+j] = matriz_t1[i*N+j];
        matriz_t1[i*N+j] = matriz_rs[i*N+j];
    }
}


// BLOQUE PRINCIPAL
    // ./wave -N 256 -T 10000 -H 12 -f archivoSalida.raw
__host__ int main (int argc, char **argv){

// INICIO: RECOLECCION DE DATOS INICIALES
    // Variables
    int c, tamanoGrilla, tamanoBloqueX, tamanoBloqueY, numPasos;
    char *charGrilla, *charBloqueX, *charBloqueY, *charNPasos, *archivoSalida;
    int parametros = 0;
    // Obtener datos ingresados en la instruccion en consola mediante getOpt, para ser almacenados respectivamente
    while ((c = getopt (argc, argv, "N:x:y:T:f:")) != -1){
        switch (c){
            // Almacena y setea el dato dependiendo del valor entregado por parametro.
            case 'N':
                charGrilla= optarg;
                tamanoGrilla = atoi(charGrilla);
                parametros ++;
                break;
            case 'x':
                charBloqueX = optarg;
                tamanoBloqueX = atoi(charBloqueX);
                parametros ++;
                break;
            case 'y':
                charBloqueY = optarg;
                tamanoBloqueY = atoi(charBloqueY);
                parametros ++;
                break;
            case 'T':
                charNPasos = optarg;
                numPasos = atoi(charNPasos);
                parametros ++;
                break; 
            case 'f':
                archivoSalida = optarg;
                parametros ++;
                break;
            case '?':
                //Si es que se ingresó un parámetro que existe pero no se entregó un argumento requerido
                if(optopt == 'N' || optopt == 'x' || optopt == 'y' || optopt == 'T' || optopt == 'f'){
                    fprintf (stderr, "El parametro -%c no puede estar vacio.\n", optopt);
                //Si es que la opción ingresada no existe
                } else if (isprint (optopt)){ 
                    fprintf (stderr, "El parametro -%c no es requerido. \n", optopt);
                }
            default:
                abort ();
        }       
    }
    //Se validan los parámetros obligatorias, validando si fueron 5
    if(parametros != 5){ 
        printf("Se deben ingresar los siguientes parametros: \n-N <Tamano de la grilla>\n-T <Numero de Pasos>\n-H <Numero de hebras>\n-f <Archivo de salida>\n");
        return 0;
    }
    //Se validan los parámetros de entrada
    if(validar_entrada(tamanoGrilla, numPasos, tamanoBloqueX, tamanoBloqueY ) == 0){
        return 0;
    }
// FIN: RECOLECCION DE DATOS INICIALES

    //Pedir memoria al host, para las matrices (CPU)
    int tamArreglo = tamanoGrilla*tamanoGrilla;
    float *h_matriz_t0 = (float *)malloc(tamArreglo*sizeof(float));
    float *h_matriz_t1 = (float *)malloc(tamArreglo*sizeof(float));
    float *h_matriz_rs = (float *)malloc(tamArreglo*sizeof(float));
    //Inicializar valores para cada matriz del host
    matriz_ceros(h_matriz_t0,tamanoGrilla);
    matriz_ceros(h_matriz_t1,tamanoGrilla);
    matriz_ceros(h_matriz_rs,tamanoGrilla);

// INICIO: SOLUCION CUDA   
    //Se empieza a medir el tiempo
    hipEvent_t tInicioGpu, tFinGpu;
    float tiempoTranscurrido;
    hipEventCreate(&tInicioGpu);
    hipEventCreate(&tFinGpu);
    hipEventRecord(tInicioGpu,0);
    time_t tInicio = time(NULL);

    //Numero de threads en cada bloquewarpsTotales
    dim3 tamBloque, tamGrilla;
    tamGrilla.x = (int)ceil((float)tamanoGrilla/tamanoBloqueX);
    tamGrilla.y = (int)ceil((float)tamanoGrilla/tamanoBloqueY);
    //Numero de bloques en la grilla
    tamBloque.x = tamanoBloqueX;
    tamBloque.y = tamanoBloqueY;

    int numBloques = tamGrilla.x*tamGrilla.y;
    int TamBloque = tamanoBloqueX*tamanoBloqueY;

    //Pedir memoria al device, para las matrices (GPU)
    float *d_matriz_t0, *d_matriz_t1, *d_matriz_rs;
    hipMalloc(&d_matriz_t0, tamArreglo*sizeof(float));
    hipMalloc(&d_matriz_t1, tamArreglo*sizeof(float));
    hipMalloc(&d_matriz_rs, tamArreglo*sizeof(float));
    //Se copia la matriz del host a la matriz del devicetInicioGpu
    hipMemcpy(d_matriz_t0, h_matriz_t0, tamArreglo*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_matriz_t1, h_matriz_t1, tamArreglo*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_matriz_rs, h_matriz_rs, tamArreglo*sizeof(float), hipMemcpyHostToDevice);

    ejecutar_schroedinger_t0cuda<<<tamGrilla,tamBloque>>>(d_matriz_t0, tamanoGrilla);
    hipMemcpy(h_matriz_t0, d_matriz_t0, tamArreglo*sizeof(float), hipMemcpyDeviceToHost);
   
    ejecutar_schroedinger_t1cuda<<<tamGrilla,tamBloque>>>(d_matriz_t0, d_matriz_t1, tamanoGrilla);
    hipMemcpy(h_matriz_t1, d_matriz_t1, tamArreglo*sizeof(float), hipMemcpyDeviceToHost);
    
    for(int i = 2; i <= numPasos; i++){
        ejecutar_schroedinger_tncuda<<<tamGrilla,tamBloque>>>(d_matriz_t0, d_matriz_t1, d_matriz_rs, tamanoGrilla);
        copiar_matriz_cuda<<<tamGrilla,tamBloque>>>(d_matriz_t0, d_matriz_t1, d_matriz_rs, tamanoGrilla);
    }
    hipMemcpy(h_matriz_t1, d_matriz_t1, tamArreglo*sizeof(float), hipMemcpyDeviceToHost);
// FIN: SOLUCION CUDA

    //Se termina de medir el tiempo
    time_t tFin = time(NULL);
    hipEventRecord(tFinGpu,0);
    hipEventSynchronize(tFinGpu);
    hipEventElapsedTime( &tiempoTranscurrido, tInicioGpu, tFinGpu);
    printf("Tiempo (wall-clock)  : %f seg.\n", (float)(tFin-tInicio));
    printf("Tiempo transcurrido  : %f seg.\n", tiempoTranscurrido/1000);

    //Warps Utilizados
    int device, warpsActivos, warpsTotales;
    //Obtener propiedades de la GPU
    hipDeviceProp_t propiedadesDevice;
    hipGetDevice(&device);
    hipGetDeviceProperties(&propiedadesDevice, device);
    //Porcentaje de warps utilizados
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBloques, ejecutar_schroedinger_t0cuda, TamBloque, 0);
    warpsActivos = numBloques*TamBloque /propiedadesDevice.warpSize;
    warpsTotales = propiedadesDevice.maxThreadsPerMultiProcessor/propiedadesDevice.warpSize;
    printf ("Warps totales        : %d uds.\n", warpsTotales);
    printf ("Warps activos        : %d uds.\n", warpsActivos);
    printf ("Porcentaje utilizado : %f .\n", (float)warpsActivos/warpsTotales*100);

    //Se almacena la imagen
    FILE *salida2 = fopen(archivoSalida, "w");
    fwrite(h_matriz_t1, sizeof(float), tamArreglo, salida2);
    fclose(salida2);

    //Se libera toda la memoria reservada utilizada.
    hipEventDestroy(tInicioGpu);
    hipEventDestroy(tFinGpu);
    hipFree(d_matriz_t0);
    hipFree(d_matriz_t1);
    hipFree(d_matriz_rs);
    free(h_matriz_t0);
    free(h_matriz_t1);
    free(h_matriz_rs);
    return 0;
}